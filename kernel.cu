#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <random>
#include <algorithm>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t scanWithCuda(int * ret, const int * in, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__device__ inline void roundingHelper(int* left, int* right)
{
    const int binMax = 1024;
    auto l = *left;
    auto r = *right;
    if ((l % binMax + r % binMax) <= binMax)
        return;
    *left = (((l - 1) / binMax) + 1) * binMax;
    *right = (((r - 1) / binMax) + 1) * binMax;
}

__global__ void divideScanResult(int* ret, int* in) 
{
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    ret[i] = (in[i] - 1) / 1024;
    //ret[i] = in[i];

}

//could be a call on 512 threads
__global__ void scanKernel(int* cumsum, int* data) {
    auto i = threadIdx.x + blockDim.x * blockIdx.x;
    auto li = threadIdx.x;
    if(li > 1024)
        return;
    // move to shared memory per block
    __shared__ int s[1024];
    s[li] = data[i];

    __syncthreads();

    // mathematical indexing
    auto ni = li + 1;
    // upsweep
    #pragma unroll
    for (auto t = 1; t <= 10; t++)
    {
        auto shift = 1 << t-1;
        if (ni % (1 << t) == 0)
        {
            s[ni - 1] += s[ni - shift - 1];
        }
        __syncthreads();
    }


    // downsweep
    #pragma unroll
    for (auto t = 10; t > 0; t--)
    {
        auto shift = 1 << t - 1;
        // last index when the addition is not possible (it is known to be the the last index only affected)
        if (ni != 1024 && ni % (1 << t) == 0)
        {
            s[ni + shift - 1] += s[ni - 1];
        }
        __syncthreads();
    }

    cumsum[i] = s[li];
    __syncthreads();
}

//collect previous sums to have full cumulative value  
__global__ void scanPartialResults(int* sum,int* data) {
    auto i = threadIdx.x + blockDim.x * blockIdx.x;
    auto li = threadIdx.x;
    auto ni = li + 1;

    __shared__ int s[1024];
    s[li] = 0;
    // copy previous sum values to shared
    if (li < blockIdx.x)
        s[li] = data[blockDim.x * li + 1023];
    __syncthreads();

    // upsweep
    //#pragma unroll
    for (auto t = 1; t <= 10; t++)
    {
        auto shift = 1 << t - 1;
        if (ni % (1 << t) == 0)
        {
            s[ni - 1] += s[ni - shift - 1];
        }
        __syncthreads();
    }

    //add cumulative sum
    sum[i] = data[i] + s[1023];
    __syncthreads();
}

//could be a call on 512 threads
__global__ void scanKernelModified(int* cumsum, int* data) {
    auto i = threadIdx.x + blockDim.x * blockIdx.x;
    auto li = threadIdx.x;
    if (li > 1024)
        return;
    // move to shared memory per block
    __shared__ int s[1024];
    s[li] = data[i];

    __syncthreads();

    // mathematical indexing
    auto ni = li + 1;
    // upsweep
#pragma unroll
    for (auto t = 1; t <= 10; t++)
    {
        auto shift = 1 << t - 1;
        if (ni % (1 << t) == 0)
        {
            roundingHelper(&s[li-shift], &s[li]);
            s[ni - 1] += s[ni - shift - 1];
        }
        __syncthreads();
    }


    // downsweep
    #pragma unroll
    for (auto t = 10; t > 0; t--)
    {
        auto shift = 1 << t - 1;
        // last index when the addition is not possible (it is known to be the the last index only affected)
        if (ni != 1024 && ni % (1 << t) == 0)
        {
            s[ni + shift - 1] += s[ni - 1];
        }
        __syncthreads();
    }
    cumsum[i] = s[li];
    __syncthreads();
}

//collect previous sums to have full cumulative value  
__global__ void scanPartialResultsModified(int* sum, int* data) {
    auto i = threadIdx.x + blockDim.x * blockIdx.x;
    auto li = threadIdx.x;
    auto ni = li + 1;

    __shared__ int s[1024];
    s[li] = 0;
    // copy previous sum values to shared
    if (li < blockIdx.x)
        s[li] = data[blockDim.x * li + 1023];
    __syncthreads();

    // upsweep
    //#pragma unroll
    for (auto t = 1; t <= 10; t++)
    {
        auto shift = 1 << t - 1;
        if (ni % (1 << t) == 0)
        {
            roundingHelper(&s[li - shift], &s[li]);
            s[ni - 1] += s[ni - shift - 1];
        }
        __syncthreads();
    }

    //add cumulative sum
    sum[i] = data[i] + s[1023];
    __syncthreads();
}

int main()
{
    const int arraySize = 1024*1024;
    int* a = new int[arraySize];
    int* b = new int[arraySize];

    std::random_device rd;
    auto gen = std::mt19937(2502341);
    auto distribution = std::binomial_distribution<int>(1023, 1. / 128.);

    std::generate(a, a + arraySize, [&]() {
        return distribution(gen);
        });

    // Add vectors in parallel.
    hipError_t cudaStatus = scanWithCuda(b, a, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    std::cout << "test full sum" << std::endl;
    for (auto i = 0; i < 40; i++)
        std::cout << a[i] << ',';
    std::cout<<std::endl;
    for (auto i = 10; i-- > 0;)
        std::cout << '-';
    std::cout << std::endl;
    for (auto j = 0; j < 40; j++)
        std::cout << b[j] << ',';
    std::cout << std::endl;
    for (auto i = 10; i-- > 0;)
        std::cout << '-';
    std::cout << std::endl;
    for (auto j = 1010; j < 1040; j++)
        std::cout << b[j] << ',';

    auto acc = 0;
    auto l = 0;
    for (auto i = 0; i < arraySize; i++)
    {
        if (l != b[i])
        {
            l = b[i];
            acc = 0;
            continue;
        }
        acc += a[i];
        if (acc > 1024)
            std::cout << "Error at index " << i << std::endl;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t scanWithCuda(int * ret, const int * in, unsigned int size)
{
    int* dev_ret = 0;
    int* dev_in = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_ret, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_in, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_in, in, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    // Launch a kernel on the GPU with one thread for each element.
    scanKernelModified <<<1024, 1024 >> > (dev_ret, dev_in);
    // sync blocks
    // sum over blocks
    scanPartialResults<<<1024, 1024>>>(dev_in, dev_ret);
    divideScanResult<< <1024, 1024 >> > (dev_ret, dev_in);



    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(ret, dev_ret, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_ret);
    hipFree(dev_in);

    return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
